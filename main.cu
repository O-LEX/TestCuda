#include <iostream>
#include <hip/hip_runtime.h>

// CUDAカーネル関数: 2つの整数を足し算
__global__ void addKernel(int a, int b, int *result) {
    *result = a + b;
}

// ホスト関数: GPUで加算を実行する
int addGPU(int a, int b) {
    int *d_result;    // デバイスメモリ用のポインタ
    int h_result;     // ホストメモリ用の変数

    // デバイスメモリの確保
    hipMalloc((void**)&d_result, sizeof(int));

    // カーネルの呼び出し (1つのスレッドで実行)
    addKernel<<<1, 1>>>(a, b, d_result);

    // エラー処理
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // デバイスからホストへ結果をコピー
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // デバイスメモリを解放
    hipFree(d_result);

    return h_result;
}
